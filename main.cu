#include "hip/hip_runtime.h"
﻿     
#include "hip/hip_runtime.h"
#include ""

#include <chrono>
#include <stdio.h>
#include <iostream>

const int size = 1024;//works only for powers of 2, need to repair later
double results[6];//an array to compare if the results were correct

void printMatrix(float **matrix) {
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            std::cout << matrix[i][j] << "  |";
        }
        std::cout << std::endl;
    }
}

void printCudaMatrix(float* matrix)
{
    long int cudaSize = size * size;
    for (int i = 0; i < cudaSIZE; i++)
        std::cout << matrix[i] << "   |";
}


__global__ void mult(float* a, float* b, float* c)
{
    int j = threadIdx.x + blockIdx.x * blockDim.x;//col
    int i = threadIdx.y + blockIdx.y * blockDim.y;//row
    float sum=0;
    for (int shift = 0; shift < size; shift++)
        sum += a[i * size + shift] * b[shift * size + j];
    
    c[i*size+j] = sum;
}


hipError_t CudaMultiplyMatrixes(float **aORG, float **bORG)
{
    float* a = (float*)malloc(size * size * sizeof(float));
    for (int i = 0; i < size; ++i)
        for (int j = 0; j < size; ++j)
            a[i * size + j] = aORG[i][j];
    float* b = (float*)malloc(size * size * sizeof(float));
    for (int i = 0; i < size; ++i)
        for (int j = 0; j < size; ++j)
            b[i * size + j] = bORG[i][j];

    float* c = (float*)malloc(size * size * sizeof(float));

    for (int i = 0; i < size*size; i++)
            c[i] = 0;

    float* cudaMatrixA = 0;
    float* cudaMatrixB = 0;
    float* cudaMatrixC = 0;
    hipError_t cudaStatus;

    //set gpu
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    //cudaMallocs with pitch for variables
    size_t pitch;
    cudaStatus = hipMallocPitch(&cudaMatrixA, &pitch, size * sizeof(float), size);
    if (cudaStatus != hipSuccess) 
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    size_t pitch2;
    cudaStatus = hipMallocPitch(&cudaMatrixB, &pitch2, size * sizeof(float), size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    size_t pitch3;
    cudaStatus = hipMallocPitch(&cudaMatrixC, &pitch3, size * sizeof(float), size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }



    //copy arrays to GPU
    cudaStatus = hipMemcpy2D(cudaMatrixA, pitch, a, size * sizeof(float), size * sizeof(float), size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy2D(cudaMatrixB, pitch2, b, size * sizeof(float), size * sizeof(float), size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy2D(cudaMatrixC, pitch3, c, size * sizeof(float), size * sizeof(float), size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //calculate
    dim3 grid(ceilf(size / (float)32), ceilf(size / (float)32), 1);
    dim3 block(32, 32, 1);

    auto start = std::chrono::high_resolution_clock::now();
    mult << < grid,block >> > (cudaMatrixA, cudaMatrixB, cudaMatrixC);
    auto end = std::chrono::high_resolution_clock::now();


    //check for errors
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, " launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    //gpu->cpu
    cudaStatus = hipMemcpy2D(c, size*sizeof(float),cudaMatrixC, pitch3, size * sizeof(float), size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    printf("\n\n\n\n\n\n\n\n\n\n\n");
    printCudaMatrix(c);
    for (int i = 0; i < 3; i++)
        results[i+3] = c[i];
    printf("time: %d microS\n", std::chrono::duration_cast<std::chrono::microseconds>(end - start).count());
Error:
    hipFree(cudaMatrixA);
    hipFree(cudaMatrixB);
    hipFree(cudaMatrixC);

    return cudaStatus;
}


void multiplyMatrixes(float** a, float** b)
{
    float** c = (float**)malloc(size * sizeof(float*));
    
    for (int i = 0; i < size; ++i) 
        c[i] = (float*)malloc(size * sizeof(float));

    for (int i = 0; i < size; i++) 
        for (int j = 0; j < size; j++) 
            c[i][j] = 0;

    auto start = std::chrono::high_resolution_clock::now();
        for (int i = 0; i < size; i++)
            for (int j = 0; j < size; j++)
                for (int k = 0; k < size; ++k)
                    c[i][j] += a[i][k] * b[k][j];
    
    auto end = std::chrono::high_resolution_clock::now();
    //printMatrix(c);
    for (int i = 0; i < 3; i++)
        results[i] = c[0][i];
    printf("time: %d microS\n", std::chrono::duration_cast<std::chrono::microseconds>(end - start).count());

}





int main()
{
    float **a = (float**)malloc(size*sizeof(float*));
    float **b = (float**)malloc(size*sizeof(float*));
   
    for (int i = 0; i < size; i++) 
    {
        a[i] = (float*)malloc(size * sizeof(float));
        b[i] = (float*)malloc(size * sizeof(float));
    }

    for (int i = 0; i < size; i++) 
        for (int j = 0; j < size; j++) 
        {
            a[i][j] = 0;
            b[i][j] = 0;
        }
    
    
    srand(time(0));
    
    for (int i = 0; i < size; ++i)
        for (int j = 0; j < size; ++j) 
        {
            a[i][j] = static_cast<float>(std::rand()) / RAND_MAX;
            b[i][j] = static_cast<float>(std::rand()) / RAND_MAX;
        }
    
    
    multiplyMatrixes(a, b);
    CudaMultiplyMatrixes(a, b);
    for (int i = 0; i < 3; i++)
        printf("results[%d] = %f\nresults[%d] = %f\n", i, results[i], i+3, results[i+3]);
  
    /* //just a fun code to see the possibilities of CUDA
    int deviceCount, device;
    int gpuDeviceCount = 0;
    struct hipDeviceProp_t properties;
    hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
    if (cudaResultCode != hipSuccess)
        deviceCount = 0;
    for (device = 0; device < deviceCount; ++device) {
        hipGetDeviceProperties(&properties, device);
            if (device == 0)
            {
                printf("maxthreads per block %d\n", properties.maxThreadsPerBlock);
                
            }
    }
    */
    return 0;
}

